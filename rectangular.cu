#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include <hip/hip_runtime.h>
#include <time.h>
#include <vector>
#include <opencv2/opencv.hpp>
#include "opencv2/highgui.hpp"
#include <iostream>
#include <math.h>
#include <chrono>

#define K 11             // Legendre polynomial 在-1和1附近震荡比较厉害，所以K比较小时，图像复现效果差一些
#define ORD 1024         // Legendre polynomial的阶数order，最好是32的倍数
#define TILE_WIDTH 32   // block的宽

static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number)
{
  if(err!=hipSuccess)
  {
    fprintf(stderr,"%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n",msg,file_name,line_number,hipGetErrorString(err));
    std::cin.get();
    exit(EXIT_FAILURE);
  }
}

#define SAFE_CALL(call,msg) _safe_cuda_call((call),(msg),__FILE__,__LINE__)

// bilinear interpolation, enlarge k times in x and y direction
__global__ void inter_liner_k(float *dataOut, uchar *dataIn, int imgHeight, int imgWidth, int imgHeight_k, int imgWidth_k, float scale)
{   
  int xIdx = threadIdx.x + blockIdx.x * blockDim.x;	
  int yIdx = threadIdx.y + blockIdx.y * blockDim.y;

  if(xIdx < imgWidth_k && yIdx < imgHeight_k)
  {        
    float fx = (float)((xIdx + 0.5f) * scale - 0.5f);
    int sx = floorf(fx);
    fx -= sx;
    sx = min(sx, imgWidth - 1);
    int sx2 = min(sx + 1, imgWidth - 1);
    if(sx < 0)
      sx2 = 0, sx = 0;

    float2 cbufx;
    cbufx.x = 1.f - fx;
    cbufx.y = fx;

    float fy = (float)((yIdx + 0.5f) * scale - 0.5f);
    int sy = floorf(fy);
    fy -= sy;
    sy = min(sy, imgHeight - 1);
    int sy2 = min(sy + 1, imgHeight - 1);
    if(sy < 0)
        sy2 = 0, sy = 0;

    float2 cbufy;
    cbufy.x = 1.f - fy;
    cbufy.y = fy;

    uchar s11 = dataIn[sy * imgWidth + sx];
    uchar s12 = dataIn[sy * imgWidth + sx2];
    uchar s21 = dataIn[sy2 * imgWidth + sx];
    uchar s22 = dataIn[sy2 * imgWidth + sx2];
       
    float h_rst00x, h_rst01x;
    h_rst00x = s11 * cbufx.x + s12 * cbufx.y;
    h_rst01x = s21 * cbufx.x + s22 * cbufx.y;

    dataOut[yIdx*imgWidth_k + xIdx] = (h_rst00x * cbufy.x + h_rst01x * cbufy.y) / 127.5f - 1.f; 
  }
}

__constant__ float dj[ORD];
__constant__ float j2_1[ORD]; //2 * j -1
__constant__ float j_1[ORD]; // j - 1
// v:勒让德多项式的值； W：把[-1, 1]等分为W份； div：每一份的长度
// 生成的多项式是(Row) * (Col): ORD * W 的矩阵
__global__ void p_polynomial(float *v, const int W, const float div, const float div1)
{
  int xIdx = threadIdx.x + blockIdx.x * blockDim.x;	
  
  if(xIdx < W)
  {
    v[xIdx] = 1.0f;
    float temp_x = div * xIdx - 1.0f + div1; 
    v[xIdx + W] = temp_x;
    float p0 = 1.0f;
    float p1 = temp_x;
    for(int j=2; j<ORD; j++)
    { 
      float temp_v = v[xIdx + j * W] = (j2_1[j] * temp_x * p1 - j_1[j] * p0) * dj[j];
      p0 = p1;
      p1 = temp_v;
    }
  }
}

// https://developer.nvidia.com/blog/efficient-matrix-transpose-cuda-cc/
__global__ void matrix_transpose(float *odata, const float *idata, int matrixWidth, int matrixHeight)
{
  __shared__ float tile[TILE_WIDTH][TILE_WIDTH+1];

  int x = blockIdx.x * TILE_WIDTH + threadIdx.x;
  int y = blockIdx.y * TILE_WIDTH + threadIdx.y;
  int width = matrixWidth;

  if(x<matrixWidth && y<matrixHeight)
    tile[threadIdx.y][threadIdx.x] = idata[y*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_WIDTH + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_WIDTH + threadIdx.y;
  width = matrixHeight;

  if(x<matrixHeight && y<matrixWidth)
    odata[y*width + x] = tile[threadIdx.x][threadIdx.y];        
}

// 根据公式(48)，乘以相应系数
__global__ void multiply_coff(float *lambda_coff, float *lambda, int imgHeight_k, int imgWidth_k)
{
  int bx = blockIdx.x; int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;

  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx; 

  // double
  if(Row+Col <= ORD)
    lambda_coff[Row*ORD + Col] = lambda[Row*ORD + Col] * (float)((2*Row + 1) * (2*Col + 1)) / (float)(imgHeight_k * imgWidth_k);
  else
    lambda_coff[Row*ORD + Col] = 0.f;
}

__global__ void recon_img_f2u(uchar *recon_img, float *recon_img_float, int imgHeight, int imgWidth)
{
  int bx = blockIdx.x; int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;

  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx; 

  if((Row<imgHeight) && (Col<imgWidth))
  {
    recon_img[Row*imgWidth + Col] = (recon_img_float[Row*imgWidth + Col] + 1.f) * 127.5f;
  }  
}

int main(void)
{  
  cv::Mat img_ori = cv::imread("test_images/lenna-1024.tif", 0); 
  //////////////////////////////////////////////////// 计算 image moments /////////////////////////////////////////////////////
  // 1、计算X和Y方向的勒让德多项式；
  // 2、把原图的每个像素分成k*k个小方格(图像双线性resize)，使得积分计算更准确
  // 3、转置X方向的勒让德多项式，得到C^T
  // 4、分别计算AC^T和B*(AC^T)，得到图像矩lambda
  float scale = 1.f / (float) K;
  const int imgWidth = img_ori.cols;
  const int imgHeight = img_ori.rows;
  const int imgWidth_k = imgWidth * K;
  const int imgHeight_k = imgHeight * K;
  const int total_ph_X = std::ceil((float)imgWidth_k / (float)TILE_WIDTH);
  const int total_ph_Y = std::ceil((float)imgHeight_k / (float)TILE_WIDTH);

  float *p_in_Xdir_k, *p_in_Ydir_k, *p_in_Xdir, *p_in_Ydir; // 勒让德多项式在X和Y方向的值
  SAFE_CALL(hipMalloc((void**)&p_in_Xdir_k, ORD * imgWidth_k * sizeof(float)), "hipMalloc p_in_Xdir_k failed");  // 计算image moments
  SAFE_CALL(hipMalloc((void**)&p_in_Ydir_k, ORD * imgHeight_k * sizeof(float)), "hipMalloc p_in_Ydir_k failed");  
  SAFE_CALL(hipMalloc((void**)&p_in_Xdir, ORD * imgWidth * sizeof(float)), "hipMalloc p_in_Xdir failed");    // 重建图像
  SAFE_CALL(hipMalloc((void**)&p_in_Ydir, ORD * imgHeight * sizeof(float)), "hipMalloc p_in_Ydir failed"); 
  dim3 blockDim_P(32, 1, 1);
  dim3 gridDim_p_X_k((imgWidth_k + blockDim_P.x - 1) / blockDim_P.x, 1, 1);
  dim3 gridDim_p_Y_k((imgHeight_k + blockDim_P.x - 1) / blockDim_P.x, 1, 1);
  dim3 gridDim_p_X((imgWidth + blockDim_P.x - 1) / blockDim_P.x, 1, 1);
  dim3 gridDim_p_Y((imgHeight + blockDim_P.x - 1) / blockDim_P.x, 1, 1);

  uchar *oriImg;
  float *resImg;
  SAFE_CALL(hipMalloc((void**)&oriImg, imgHeight * imgWidth * sizeof(uchar)), "hipMalloc oriImg failed");
  SAFE_CALL(hipMalloc((void**)&resImg, imgHeight_k * imgWidth_k * sizeof(float)), "hipMalloc resImg failed");
  SAFE_CALL(hipMemcpy(oriImg, img_ori.data, imgHeight * imgWidth * sizeof(uchar), hipMemcpyHostToDevice), "oriImg hipMemcpyHostToDevice failed");
  
  hipEvent_t start, stop;
  float runtime;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  float *temp_dj = new float[ORD];
  temp_dj[0] = 1.0f;
  temp_dj[1] = 1.0f;
  float *temp_j2_1 = new float[ORD];
  temp_j2_1[0] = 1.0f;
  temp_j2_1[1] = 1.0f;
  float *temp_j_1 = new float[ORD];
  temp_j_1[0] = 1.0f;
  temp_j_1[1] = 1.0f;
  for(int j=2; j<ORD; j++)
  {
    temp_dj[j] = (float)(1) / (float)(j);
    temp_j2_1[j] = (float) (2 * j -1);
    temp_j_1[j] = (float) (j - 1);
  }

  const float dx_k = (float) (2) / (float) (imgWidth_k);
  const float dx_k1 = dx_k / 2.f;
  const float dy_k = (float) (2) / (float) (imgHeight_k);
  const float dy_k1 = dy_k / 2.f;
  const float dx = (float) (2) / (float) (imgWidth);
  const float dx1 = dx / 2.f;
  const float dy = (float) (2) / (float) (imgHeight);
  const float dy1 = dy / 2.f;

  SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dj), temp_dj, ORD * sizeof(float)), "hipMemcpyToSymbol dj failed");
  SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(j2_1), temp_j2_1, ORD * sizeof(float)), "hipMemcpyToSymbol j2_1 failed");
  SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(j_1), temp_j_1, ORD * sizeof(float)), "hipMemcpyToSymbol j_1 failed");
  p_polynomial<<<gridDim_p_X_k, blockDim_P>>>(p_in_Xdir_k, imgWidth_k, dx_k, dx_k1);  // 把X方向转置
  p_polynomial<<<gridDim_p_Y_k, blockDim_P>>>(p_in_Ydir_k, imgHeight_k, dy_k, dy_k1);
  p_polynomial<<<gridDim_p_X, blockDim_P>>>(p_in_Xdir, imgWidth, dx, dx1);   // 把X方向转置
  p_polynomial<<<gridDim_p_Y, blockDim_P>>>(p_in_Ydir, imgHeight, dy, dy1);

  dim3 blockDim_trans(32, 32);
  float *p_in_Xdir_trans_k;
  SAFE_CALL(hipMalloc((void**)&p_in_Xdir_trans_k, ORD * imgWidth_k * sizeof(float)), "hipMalloc p_in_Xdir_trans_k failed");   
  dim3 gridDim_trans_k((imgWidth_k + blockDim_trans.x - 1) / blockDim_trans.x, (ORD + blockDim_trans.y - 1) / blockDim_trans.y);
  matrix_transpose<<<gridDim_trans_k, blockDim_trans>>>(p_in_Xdir_trans_k, p_in_Xdir_k, imgWidth_k, ORD);

  dim3 blockDim_resize(32, 32);
  dim3 gridDim_resize((imgWidth_k + blockDim_resize.x - 1) / blockDim_resize.x, (imgHeight_k + blockDim_resize.y - 1) / blockDim_resize.y);
  inter_liner_k<<<gridDim_resize, blockDim_resize>>>(resImg, oriImg, imgHeight, imgWidth, imgHeight_k, imgWidth_k, scale);
  
  hipblasHandle_t handle;
  hipblasStatus_t status = hipblasCreate(&handle);
      
  if (status != HIPBLAS_STATUS_SUCCESS)
  {
    if (status == HIPBLAS_STATUS_NOT_INITIALIZED) {
      std::cout << "CUBLAS 对象实例化出错" << std::endl;
    }
    getchar();
    return EXIT_FAILURE;
  }

  float *AC;
  SAFE_CALL(hipMalloc((void**)&AC, imgHeight_k * ORD * sizeof(float)), "hipMalloc AC failed");
  const float a = 1.0f, b = 0.0f;
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, ORD, imgHeight_k, imgWidth_k, &a, 
              p_in_Xdir_trans_k, ORD, resImg, imgWidth_k, &b, AC, ORD);

  // p_in_Xdir_k 用cublas内置函数转置
  // hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, ORD, imgHeight_k, imgWidth_k, &a, 
  //           p_in_Xdir_k, ORD, resImg, imgWidth_k, &b, AC, ORD);

  float *lambda;
  float *lambda_cpu = new float[ORD * ORD];
  SAFE_CALL(hipMalloc((void**)&lambda, ORD * ORD * sizeof(float)), "hipMalloc lambda failed");
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, ORD, ORD, imgHeight_k, &a, 
              AC, ORD, p_in_Ydir_k, imgHeight_k, &b, lambda, ORD);

  // SAFE_CALL(hipMemcpy(lambda_cpu, lambda, ORD * ORD * sizeof(float), hipMemcpyDeviceToHost), "lambda to lambda_cpu failed");
  // for(int i=0; i<ORD; i++)
  // {
  //   for(int j=0; j<i; j++) 
  //   { 
  //     std::cout << lambda_cpu[(i-j)*ORD + j] * (float)((2*(i-j)+1)*(2*j+1)) / (float)(imgHeight_k*imgWidth_k) <<" ";
  //   }
  //   std::cout << std::endl;
  // }

  //////////////////////////////////////////////////// 重建图像 /////////////////////////////////////////////////////
  float *p_in_Ydir_trans;
  SAFE_CALL(hipMalloc((void**)&p_in_Ydir_trans, ORD * imgHeight * sizeof(float)), "hipMalloc p_in_Ydir_trans failed"); 
  dim3 gridDim_trans((imgHeight + blockDim_trans.x - 1) / blockDim_trans.x, (ORD + blockDim_trans.y - 1) / blockDim_trans.y);
  matrix_transpose<<<gridDim_trans, blockDim_trans>>>(p_in_Ydir_trans, p_in_Ydir, imgHeight, ORD);

  float *lambda_coff;
  SAFE_CALL(hipMalloc((void**)&lambda_coff, ORD * ORD * sizeof(float)), "hipMalloc lambda_coff failed");
  dim3 blockDim_coff(32, 32);
  dim3 gridDim_coff((ORD + blockDim_coff.x-1) / blockDim_coff.x, (ORD + blockDim_coff.y-1) / blockDim_coff.y);
  multiply_coff<<<gridDim_coff, blockDim_coff>>>(lambda_coff, lambda, imgHeight_k, imgWidth_k);

  float *recon_img_float, *recon_img_tmp;
  SAFE_CALL(hipMalloc((void**)&recon_img_float, imgWidth * imgHeight * sizeof(float)), "hipMalloc recon_img_float failed");
  SAFE_CALL(hipMalloc((void**)&recon_img_tmp, ORD * imgWidth * sizeof(float)), "hipMalloc recon_img_tmp failed");
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, imgWidth, ORD, ORD, &a, 
              p_in_Xdir, imgWidth, lambda_coff, ORD, &b, recon_img_tmp, imgWidth);
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, imgWidth, imgHeight, ORD, &a,
              recon_img_tmp, imgWidth, p_in_Ydir_trans, ORD, &b, recon_img_float, imgWidth);
  
  // 把float型的重建图像转换成uchar
  uchar *recon_img;
  SAFE_CALL(hipMalloc((void**)&recon_img, imgWidth * imgHeight * sizeof(uchar)), "hipMalloc recon_img failed");
  dim3 gridDim_f2u((imgWidth + blockDim_coff.x-1) / blockDim_coff.x, (imgHeight + blockDim_coff.y-1) / blockDim_coff.y);
  recon_img_f2u<<<gridDim_f2u, blockDim_coff>>>(recon_img, recon_img_float, imgHeight, imgWidth);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("CUDA Error: %s\n", hipGetErrorString(err));
    // Possibly: exit(-1) if program cannot continue....
  } 

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&runtime, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  std::cout << "hipEvent_t time: " << runtime * 1000 << " us" << std::endl;

  cv::Mat recon_img_cpu(imgHeight, imgWidth, CV_8UC1);
  SAFE_CALL(hipMemcpy(recon_img_cpu.data, recon_img, imgHeight*imgWidth*sizeof(uchar), hipMemcpyDeviceToHost), "hipMemcpy recon_img_cpu.data failed");
  cv::imwrite("test_images/lenna-1024_" + std::to_string(K) +"_"+ std::to_string(ORD) + "_recons.tif", recon_img_cpu);

  float MSE=0.f, PSNR=0.f;
  for (int i = 0; i < imgHeight; i++)
	{
		for (int j = 0; j < imgWidth; j++)
		{
			MSE += (img_ori.ptr<uchar>(i)[j] - recon_img_cpu.ptr<uchar>(i)[j])*(img_ori.ptr<uchar>(i)[j] - recon_img_cpu.ptr<uchar>(i)[j]);
		}
	}
	MSE = MSE / (imgHeight * imgWidth);
  PSNR = 20 * log10(255) - 10 * log10(MSE);
  std::cout << "MSE: " << MSE << std::endl;
  std::cout << "PSNR: " << PSNR << std::endl;
  
  delete[] temp_dj;
  delete[] temp_j_1;
  delete[] temp_j2_1;
  delete[] lambda_cpu;
  // delete[] p_in_Xdir_cpu;
  // delete[] p_in_Ydir_cpu;
  SAFE_CALL(hipFree(p_in_Xdir_k), "free p_in_Xdir_k failed");
  SAFE_CALL(hipFree(p_in_Ydir_k), "free p_in_Ydir_k failed");
  SAFE_CALL(hipFree(p_in_Xdir), "free p_in_Xdir failed");
  SAFE_CALL(hipFree(p_in_Ydir), "free p_in_Ydir_k failed");
  SAFE_CALL(hipFree(p_in_Xdir_trans_k), "free p_in_Xdir_trans_k failed");
  SAFE_CALL(hipFree(p_in_Ydir_trans), "free p_in_Ydir_trans failed");
  SAFE_CALL(hipFree(AC), "free AC failed");
  SAFE_CALL(hipFree(lambda), "free lambda failed");
  SAFE_CALL(hipFree(lambda_coff), "free lambda_coff failed");
	SAFE_CALL(hipFree(oriImg), "free oriImg failed");
  SAFE_CALL(hipFree(resImg), "free resImg failed");
  SAFE_CALL(hipFree(recon_img), "free recon_img failed");
  SAFE_CALL(hipFree(recon_img_tmp), "free recon_img failed");
  SAFE_CALL(hipFree(recon_img_float), "free recon_img failed");
  return 0;
}
